#include "hip/hip_runtime.h"
/*
 * CudaSingleFilterKernel.cu
 *
 *  Created on: Nov 9, 2014
 *      Author: prabodha
 */

#ifndef CUDASINGLEFILTERKERNEL_CU_
#define CUDASINGLEFILTERKERNEL_CU_

#include "GpuEventConsumer.h"
#include "ByteBufferStructs.h"
#include "CudaSingleFilterKernel.h"
#include "Filter.h"
#include "CudaEvent.h"
#include "helper_timer.h"
#include "CudaFilterKernelCore.h"

namespace SiddhiGpu
{

__global__ void ProcessEventsSingleFilterKernel(SingleFilterKernelInput * _pInput)
{
	if(threadIdx.x >= _pInput->i_EventsPerBlock || threadIdx.y > 0 || blockIdx.y > 0)
		return;

	if((blockIdx.x == _pInput->i_EventCount / _pInput->i_EventsPerBlock) && // last thread block
			(threadIdx.x >= _pInput->i_EventCount % _pInput->i_EventsPerBlock))
	{
		return;
	}

	EventMeta * pEventMeta = (EventMeta*) (_pInput->p_ByteBuffer + _pInput->i_EventMetaPosition);
	/*__shared__*/ EventMeta mEventMeta = *pEventMeta;

	// get assigned event
	int iEventIdx = (blockIdx.x * _pInput->i_EventsPerBlock) +  threadIdx.x;
	char * pEvent = (_pInput->p_ByteBuffer + _pInput->i_EventDataPosition) + (_pInput->i_SizeOfEvent * iEventIdx);

	// get assigned filter
	/*__shared__*/ Filter mFilter = *_pInput->ap_Filter;

	// get results array
	MatchedEvents * pMatchedEvents = (MatchedEvents*) (_pInput->p_ByteBuffer + _pInput->i_ResultsPosition);

	int iCurrentNodeIdx = 0;
	bool bResult = Evaluate(mFilter, mEventMeta, pEvent, iCurrentNodeIdx);

	//TODO improve results sending
	if(bResult)
	{
		pMatchedEvents->a_ResultEvents[iEventIdx] = 1;
	}
	else // ~ possible way to avoid hipMemset from host
	{
		pMatchedEvents->a_ResultEvents[iEventIdx] = 0;
	}
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

CudaSingleFilterKernel::CudaSingleFilterKernel(int _iMaxBufferSize, GpuEventConsumer * _pConsumer, FILE * _fpLog) :
		CudaKernelBase(_pConsumer, _fpLog),
		i_MaxNumberOfEvents(_iMaxBufferSize)
{
	i_EventsPerBlock = _iMaxBufferSize / 4; // TODO: change this dynamically based on MaxBuffersize

	p_HostEventBuffer = NULL;
	i_EventBufferSize = 0;
	p_HostInput= NULL;
	p_DeviceInput = NULL;
	p_StopWatch = NULL;
	i_NumAttributes = 0;
}

CudaSingleFilterKernel::CudaSingleFilterKernel(int _iMaxBufferSize, int _iEventsPerBlock, GpuEventConsumer * _pConsumer, FILE * _fpLog) :
	CudaKernelBase(_pConsumer, _fpLog),
	i_MaxNumberOfEvents(_iMaxBufferSize)
{
	if(_iEventsPerBlock > 0)
	{
		i_EventsPerBlock = _iEventsPerBlock;
	}
	else
	{
		i_EventsPerBlock = _iMaxBufferSize / 4;
	}

	p_HostEventBuffer = NULL;
	i_EventBufferSize = 0;
	p_HostInput= NULL;
	p_DeviceInput = NULL;
	p_StopWatch = NULL;
	i_NumAttributes = 0;
}

CudaSingleFilterKernel::~CudaSingleFilterKernel()
{
	CUDA_CHECK_RETURN(hipFree(p_DeviceInput->p_ByteBuffer));
	CUDA_CHECK_RETURN(hipFree(p_DeviceInput));

	free(p_HostInput);

	CUDA_CHECK_RETURN(hipDeviceReset());

	sdkDeleteTimer(&p_StopWatch);
}

void CudaSingleFilterKernel::SetEventBuffer(char * _pBuffer, int _iSize)
{
	p_HostEventBuffer = _pBuffer;
	i_EventBufferSize = _iSize;

	p_HostInput->i_ResultsPosition = i_ResultsBufferPosition;
	p_HostInput->i_EventMetaPosition = i_EventMetaBufferPosition;
	p_HostInput->i_EventDataPosition = i_EventDataBufferPosition;
	p_HostInput->i_SizeOfEvent = i_SizeOfEvent;
	
	CUDA_CHECK_RETURN(hipMalloc((void**) &p_HostInput->p_ByteBuffer, sizeof(char) * i_EventBufferSize)); // device allocate ByteBuffer
	CUDA_CHECK_RETURN(hipPeekAtLastError());
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	fprintf(fp_Log, "CudaSingleFilterKernel EventBuffer [Ptr=%p Size=%d]\n", p_HostEventBuffer, i_EventBufferSize);
	fprintf(fp_Log, "CudaSingleFilterKernel ResultsBufferPosition   : %d\n", i_ResultsBufferPosition);
	fprintf(fp_Log, "CudaSingleFilterKernel EventMetaBufferPosition : %d\n", i_EventMetaBufferPosition);
	fprintf(fp_Log, "CudaSingleFilterKernel EventDataBufferPosition : %d\n", i_EventDataBufferPosition);
	fprintf(fp_Log, "CudaSingleFilterKernel SizeOfEvent             : %d\n", i_SizeOfEvent);
	fprintf(fp_Log, "Device byte buffer ptr : %p \n", p_HostInput->p_ByteBuffer);
}

void CudaSingleFilterKernel::Initialize()
{
	fprintf(fp_Log, "CudaSingleFilterKernel::Initialize [EventsPerBlock=%d]\n", i_EventsPerBlock);

	sdkCreateTimer(&p_StopWatch);

	p_HostInput = (SingleFilterKernelInput*) malloc(sizeof(SingleFilterKernelInput)); // host allocate Kernel input struct
	CUDA_CHECK_RETURN(hipMalloc((void**) &p_DeviceInput, sizeof(SingleFilterKernelInput))); // device allocate Kernel input struct

	p_HostInput->i_MaxEventCount = i_MaxNumberOfEvents;

	p_HostInput->i_EventsPerBlock = i_EventsPerBlock;

	CUDA_CHECK_RETURN(hipPeekAtLastError());
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

}

void CudaSingleFilterKernel::ProcessEvents(int _iNumEvents)
{
	sdkStartTimer(&p_StopWatch);

	p_HostInput->i_EventCount = _iNumEvents;


	fprintf(fp_Log, "Device byte buffer ptr : %p \n", p_HostInput->p_ByteBuffer);

	//TODO: async copy
	CUDA_CHECK_RETURN(hipMemcpy(p_HostInput->p_ByteBuffer, p_HostEventBuffer, sizeof(char) * i_EventBufferSize, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(p_DeviceInput, p_HostInput, sizeof(SingleFilterKernelInput), hipMemcpyHostToDevice));

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	// call entry kernel
	int numBlocksX = _iNumEvents / i_EventsPerBlock;
	int numBlocksY = 1;
	dim3 numBlocks = dim3(numBlocksX, numBlocksY);
	dim3 numThreads = dim3(i_EventsPerBlock, 1);

	ProcessEventsSingleFilterKernel<<<numBlocks, numThreads>>>(p_DeviceInput);
	CUDA_CHECK_RETURN(hipPeekAtLastError());
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	//fprintf(fp_Log, "[ProcessEvents] Copying back results\n");
	CUDA_CHECK_RETURN(hipMemcpy(
			p_HostEventBuffer,
			p_DeviceInput->p_ByteBuffer,
			sizeof(char) * 4 * i_MaxNumberOfEvents,
			hipMemcpyDeviceToHost));

	sdkStopTimer(&p_StopWatch);

	float fElapsed = sdkGetTimerValue(&p_StopWatch);
	fprintf(fp_Log, "[ProcessEvents] Stats : Elapsed=%f ms\n", fElapsed);
	fflush(fp_Log);

	lst_ElapsedTimes.push_back(fElapsed);

	sdkResetTimer(&p_StopWatch);
//	i_NumEvents = 0;
}

void CudaSingleFilterKernel::AddFilterToDevice(Filter * _pFilter)
{
	lst_HostFilters.push_back(_pFilter);
}

void CudaSingleFilterKernel::CopyFiltersToDevice()
{
	if(lst_HostFilters.size() > 1)
	{
		fprintf(fp_Log, "[ERROR] More than one filter defined in CudaSingleFilterKernel : FilterCount=%lu", lst_HostFilters.size());
		fprintf(fp_Log, "[ERROR] Using the first filter for processing");
	}

	CUDA_CHECK_RETURN(hipMalloc(
			(void**) &p_HostInput->ap_Filter,
			sizeof(Filter)));

	Filter * apHostFilters = (Filter *) malloc(sizeof(Filter));

	std::list<Filter*>::iterator ite = lst_HostFilters.begin();
	for(int i=0; i<1; ++i, ite++)
	{
		Filter * pFilter = *ite;

		apHostFilters[i].i_FilterId = pFilter->i_FilterId;
		apHostFilters[i].i_NodeCount = pFilter->i_NodeCount;
		apHostFilters[i].ap_ExecutorNodes = NULL;

		CUDA_CHECK_RETURN(hipMalloc(
				(void**) &apHostFilters[i].ap_ExecutorNodes,
				sizeof(ExecutorNode) * pFilter->i_NodeCount));

		CUDA_CHECK_RETURN(hipMemcpy(
				apHostFilters[i].ap_ExecutorNodes,
				pFilter->ap_ExecutorNodes,
				sizeof(ExecutorNode) * pFilter->i_NodeCount,
				hipMemcpyHostToDevice));

		delete pFilter;
	}

	CUDA_CHECK_RETURN(hipMemcpy(
			p_HostInput->ap_Filter,
			apHostFilters,
			sizeof(Filter),
			hipMemcpyHostToDevice));


	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	free(apHostFilters);
	apHostFilters = NULL;

	lst_HostFilters.clear();

}

float CudaSingleFilterKernel::GetElapsedTimeAverage()
{
	float total = 0;
	std::list<float>::iterator ite = lst_ElapsedTimes.begin();
	while(ite != lst_ElapsedTimes.end())
	{
		total += *ite;
		++ite;
	}

	return (total / lst_ElapsedTimes.size());
}

};

#endif


