#ifndef __GPU_WINDOW_EVENT_BUFFER_CU_
#define __GPU_WINDOW_EVENT_BUFFER_CU_

#include "GpuWindowEventBuffer.h"

namespace SiddhiGpu
{

GpuWindowEventBuffer::GpuWindowEventBuffer(std::string _sName, int _iDeviceId, GpuMetaEvent * _pMetaEvent, FILE * _fpLog) :
		GpuStreamEventBuffer(_sName, _iDeviceId, _pMetaEvent, _fpLog),
		i_RemainingCount(0),
		p_ReadOnlyDeviceEventBufferPtr(NULL)
{
	fprintf(fp_Log, "[GpuWindowEventBuffer] <%s> Created with device id : %d \n", _sName.c_str(), i_DeviceId);
	fflush(fp_Log);
}

GpuWindowEventBuffer::~GpuWindowEventBuffer()
{
	fprintf(fp_Log, "[GpuWindowEventBuffer] <%s> destroy\n", s_Name.c_str());
	fflush(fp_Log);


	if(p_ReadOnlyDeviceEventBufferPtr)
	{
		CUDA_CHECK_RETURN(hipFree(p_ReadOnlyDeviceEventBufferPtr));
		p_ReadOnlyDeviceEventBufferPtr = NULL;
	}
}

char * GpuWindowEventBuffer::CreateEventBuffer(int _iEventCount)
{
	i_EventCount = _iEventCount;
	i_RemainingCount = i_EventCount;

	i_EventBufferSizeInBytes = _iEventCount * p_HostMetaEvent->i_SizeOfEventInBytes;
	fprintf(fp_Log, "[GpuWindowEventBuffer] <%s> Allocating ByteBuffer for %d events (x %d) : %d bytes \n",
			s_Name.c_str(), _iEventCount, p_HostMetaEvent->i_SizeOfEventInBytes, (int)(sizeof(char) * i_EventBufferSizeInBytes));
	fflush(fp_Log);

	GpuCudaHelper::AllocateHostMemory(true, &p_UnalignedBuffer, &p_HostEventBuffer, i_EventBufferSizeInBytes, s_Name.c_str(), fp_Log);

	CUDA_CHECK_RETURN(hipMalloc((void**) &p_DeviceEventBuffer, i_EventBufferSizeInBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**) &p_ReadOnlyDeviceEventBufferPtr, i_EventBufferSizeInBytes));

	fprintf(fp_Log, "[GpuWindowEventBuffer] <%s> Host ByteBuffer [Ptr=%p Size=%d]\n", s_Name.c_str(), p_HostEventBuffer, i_EventBufferSizeInBytes);
	fprintf(fp_Log, "[GpuWindowEventBuffer] <%s> Device ReadWrite ByteBuffer [Ptr=%p] \n", s_Name.c_str(), p_DeviceEventBuffer);
	fprintf(fp_Log, "[GpuWindowEventBuffer] <%s> Device ReadOnly ByteBuffer [Ptr=%p] \n", s_Name.c_str(), p_ReadOnlyDeviceEventBufferPtr);
	fflush(fp_Log);

	int GpuMetaEventSize = sizeof(GpuKernelMetaEvent) + sizeof(GpuKernelMetaAttribute) * p_HostMetaEvent->i_AttributeCount;

	CUDA_CHECK_RETURN(hipMalloc((void**) &p_DeviceMetaEvent, GpuMetaEventSize));

	GpuKernelMetaEvent * pHostMetaEvent = (GpuKernelMetaEvent*) malloc(GpuMetaEventSize);

	pHostMetaEvent->i_StreamIndex = p_HostMetaEvent->i_StreamIndex;
	pHostMetaEvent->i_AttributeCount = p_HostMetaEvent->i_AttributeCount;
	pHostMetaEvent->i_SizeOfEventInBytes = p_HostMetaEvent->i_SizeOfEventInBytes;

	for(int i=0; i<p_HostMetaEvent->i_AttributeCount; ++i)
	{
		pHostMetaEvent->p_Attributes[i].i_Type = p_HostMetaEvent->p_Attributes[i].i_Type;
		pHostMetaEvent->p_Attributes[i].i_Position = p_HostMetaEvent->p_Attributes[i].i_Position;
		pHostMetaEvent->p_Attributes[i].i_Length = p_HostMetaEvent->p_Attributes[i].i_Length;
	}

	CUDA_CHECK_RETURN(hipMemcpy(
			p_DeviceMetaEvent,
			pHostMetaEvent,
			GpuMetaEventSize,
			hipMemcpyHostToDevice));

	CUDA_CHECK_RETURN(hipPeekAtLastError());
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	free(pHostMetaEvent);
	pHostMetaEvent = NULL;

	return p_HostEventBuffer;
}

void GpuWindowEventBuffer::Sync(int _iNumEvents, bool _bAsync)
{
#if GPU_DEBUG >= GPU_DEBUG_LEVEL_INFO
	fprintf(fp_Log, "[GpuWindowEventBuffer] <%s> Sync : Async=%d\n", s_Name.c_str(), _bAsync);
#endif

#ifdef _GLIBCXX_ATOMIC_BUILTINS

	// swap two buffers
	// priority is for readonly buffer

	char* tmp = p_ReadOnlyDeviceEventBufferPtr;
	while(!__sync_bool_compare_and_swap(
			&p_ReadOnlyDeviceEventBufferPtr,
			p_ReadOnlyDeviceEventBufferPtr,
			p_DeviceEventBuffer)){};
	while(!__sync_bool_compare_and_swap(
			&p_DeviceEventBuffer,
			p_DeviceEventBuffer,
			tmp)){};

#else
  #error no atomic operations available!
#endif

	if(_iNumEvents > i_RemainingCount)
	{
		i_RemainingCount = 0;
	}
	else
	{
		i_RemainingCount -= _iNumEvents;
	}

	if(_bAsync)
	{
		// update readwrite copy with swapped readonly buffer
		CUDA_CHECK_RETURN(hipMemcpyAsync(
				p_DeviceEventBuffer,
				p_ReadOnlyDeviceEventBufferPtr,
				i_EventBufferSizeInBytes,
				hipMemcpyDeviceToDevice));
	}
	else
	{
		// update readwrite copy with swapped readonly buffer
		CUDA_CHECK_RETURN(hipMemcpy(
				p_DeviceEventBuffer,
				p_ReadOnlyDeviceEventBufferPtr,
				i_EventBufferSizeInBytes,
				hipMemcpyDeviceToDevice));
	}
}

}

#endif

