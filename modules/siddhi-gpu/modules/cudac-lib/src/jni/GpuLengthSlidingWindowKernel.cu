#include "hip/hip_runtime.h"
#ifndef _GPU_LENGTH_SLIDING_WINDOW_KERNEL_CU__
#define _GPU_LENGTH_SLIDING_WINDOW_KERNEL_CU__

#include <stdio.h>
#include <stdlib.h>

#include "GpuMetaEvent.h"
#include "GpuProcessor.h"
#include "GpuProcessorContext.h"
#include "GpuStreamEventBuffer.h"
#include "GpuIntBuffer.h"
#include "GpuKernelDataTypes.h"
#include "GpuLengthSlidingWindowKernel.h"
#include "GpuCudaHelper.h"
#include "GpuUtils.h"

namespace SiddhiGpu
{

__global__
void ProcessEventsLengthSlidingWindow(
		char               * _pInputEventBuffer,     // original input events buffer
		GpuKernelMetaEvent * _pMetaEvent,            // Meta event for original input events
		int                  _iNumberOfEvents,       // Number of events in input buffer (matched + not matched)
		char               * _pEventWindowBuffer,    // Event window buffer
		int                  _iWindowLength,         // Length of current events window
		int                  _iRemainingCount,       // Remaining free slots in Window buffer
		char               * _pResultsBuffer,        // Resulting events buffer
		int                  _iMaxEventCount,        // used for setting results array
		int                  _iSizeOfEvent,          // Size of an event
		int                  _iEventsPerBlock        // number of events allocated per block
)
{
	// avoid out of bound threads
	if(threadIdx.x >= _iEventsPerBlock || threadIdx.y > 0 || blockIdx.y > 0)
		return;

	if((blockIdx.x == _iNumberOfEvents / _iEventsPerBlock) && // last thread block
			(threadIdx.x >= _iNumberOfEvents % _iEventsPerBlock)) // extra threads
	{
		return;
	}

	// get assigned event
	int iEventIdx = (blockIdx.x * _iEventsPerBlock) + threadIdx.x;

	// get in event starting position
	char * pInEventBuffer = _pInputEventBuffer + (_iSizeOfEvent * iEventIdx);

	// output to results buffer [expired event, in event]
	char * pResultsExpiredEventBuffer = _pResultsBuffer + (_iSizeOfEvent * iEventIdx * 2);
	char * pResultsInEventBuffer = pResultsExpiredEventBuffer + _iSizeOfEvent;

	GpuEvent * pExpiredEvent = (GpuEvent *)pResultsExpiredEventBuffer;
	// calculate in/expired event pair for this event

	if(iEventIdx >= _iRemainingCount)
	{
		if(iEventIdx < _iWindowLength)
		{
			// in window buffer
			char * pExpiredOutEventInWindowBuffer = _pEventWindowBuffer + (_iSizeOfEvent * (iEventIdx - _iRemainingCount));

			GpuEvent * pWindowEvent = (GpuEvent*) pExpiredOutEventInWindowBuffer;
			if(pWindowEvent->i_Type != GpuEvent::NONE) // if window event is filled
			{
				memcpy(pResultsExpiredEventBuffer, pExpiredOutEventInWindowBuffer, _iSizeOfEvent);
				pExpiredEvent->i_Type = GpuEvent::EXPIRED;
			}
			else
			{
				memset(pResultsExpiredEventBuffer, 0, _iSizeOfEvent);
				pExpiredEvent->i_Type = GpuEvent::NONE;
			}
		}
		else
		{
			// in input event buffer
			char * pExpiredOutEventInInputBuffer = _pInputEventBuffer + (_iSizeOfEvent * (iEventIdx - _iWindowLength));

			memcpy(pResultsExpiredEventBuffer, pExpiredOutEventInInputBuffer, _iSizeOfEvent);
			pExpiredEvent->i_Type = GpuEvent::EXPIRED;
		}
	}
	else
	{
		// [NULL,inEvent]
		memset(pResultsExpiredEventBuffer, 0, _iSizeOfEvent);
		pExpiredEvent->i_Type = GpuEvent::NONE;

	}

	memcpy(pResultsInEventBuffer, pInEventBuffer, _iSizeOfEvent);

}

__global__
void ProcessEventsLengthSlidingWindowFilter(
		char               * _pInputEventBuffer,     // original input events buffer
		GpuKernelMetaEvent * _pMetaEvent,            // Meta event for original input events
		int                  _iNumberOfEvents,       // Number of events in input buffer (matched + not matched)
		int                * _pFilterdEventsIndexes, // Matched event indexes from filter kernel
		char               * _pEventWindowBuffer,    // Event window buffer
		int                  _iWindowLength,         // Length of current events window
		int                  _iRemainingCount,       // Remaining free slots in Window buffer
		char               * _pResultsBuffer,        // Resulting events buffer
		int                  _iMaxEventCount,        // used for setting results array
		int                  _iSizeOfEvent,          // Size of an event
		int                  _iEventsPerBlock        // number of events allocated per block
)
{
	// avoid out of bound threads
	if(threadIdx.x >= _iEventsPerBlock || threadIdx.y > 0 || blockIdx.y > 0)
		return;

	if((blockIdx.x == _iNumberOfEvents / _iEventsPerBlock) && // last thread block
			(threadIdx.x >= _iNumberOfEvents % _iEventsPerBlock)) // extra threads
	{
		return;
	}

	// get assigned event
	int iEventIdx = (blockIdx.x * _iEventsPerBlock) + threadIdx.x;

	// get in event starting position
	char * pInEventBuffer = _pInputEventBuffer + (_iSizeOfEvent * iEventIdx);

	// output to results buffer [expired event, in event]
	char * pResultsExpiredEventBuffer = _pResultsBuffer + (_iSizeOfEvent * iEventIdx * 2);
	char * pResultsInEventBuffer = pResultsExpiredEventBuffer + _iSizeOfEvent;

	GpuEvent * pExpiredEvent = (GpuEvent *)pResultsExpiredEventBuffer;

	// check if event in my index is a matched event
	if(_pFilterdEventsIndexes[iEventIdx] < 0)
	{
		memset(pResultsExpiredEventBuffer, 0, _iSizeOfEvent);
		pExpiredEvent->i_Type = GpuEvent::NONE;

		memset(pResultsInEventBuffer, 0, _iSizeOfEvent);
		GpuEvent * pResultsInEvent = (GpuEvent *)pResultsInEventBuffer;
		pResultsInEvent->i_Type = GpuEvent::NONE;

		return; // not matched
	}

	// calculate in/expired event pair for this event

	if(iEventIdx >= _iRemainingCount)
	{
		if(iEventIdx < _iWindowLength)
		{
			// in window buffer
			char * pExpiredOutEventInWindowBuffer = _pEventWindowBuffer + (_iSizeOfEvent * (iEventIdx - _iRemainingCount));

			GpuEvent * pWindowEvent = (GpuEvent*) pExpiredOutEventInWindowBuffer;
			if(pWindowEvent->i_Type != GpuEvent::NONE) // if window event is filled
			{
				memcpy(pResultsExpiredEventBuffer, pExpiredOutEventInWindowBuffer, _iSizeOfEvent);
				pExpiredEvent->i_Type = GpuEvent::EXPIRED;
			}
			else
			{
				memset(pResultsExpiredEventBuffer, 0, _iSizeOfEvent);
				pExpiredEvent->i_Type = GpuEvent::NONE;
			}
		}
		else
		{
			// in input event buffer
			char * pExpiredOutEventInInputBuffer = _pInputEventBuffer + (_iSizeOfEvent * (iEventIdx - _iWindowLength));

			memcpy(pResultsExpiredEventBuffer, pExpiredOutEventInInputBuffer, _iSizeOfEvent);
			pExpiredEvent->i_Type = GpuEvent::EXPIRED;
		}
	}
	else
	{
		// [NULL,inEvent]
		memset(pResultsExpiredEventBuffer, 0, _iSizeOfEvent);
		pExpiredEvent->i_Type = GpuEvent::NONE;

	}

	memcpy(pResultsInEventBuffer, pInEventBuffer, _iSizeOfEvent);
}

__device__
__forceinline__
void MoveEvent(
		int                  _iDestination,       // Position in Window buffer to move source event
		char               * _pSourceEvent,       // Source event buffer
		char               * _pEventWindowBuffer, // Window data buffer
		int                  _iSizeOfEvent,       // Size of an event
		int                  _iShift              // Offset of next event
)
{
	// get event in destination position in window
	char * pDestinationEventBuffer = _pEventWindowBuffer + (_iSizeOfEvent * _iDestination);
	GpuEvent * pDestinationEvent = (GpuEvent*) pDestinationEventBuffer;

	if(pDestinationEvent->i_Type != GpuEvent::NONE) // there is an event in destination position
	{
		// move it to next position first
		int iNextPosition = _iDestination - _iShift;
		if(iNextPosition >= 0)
		{
			MoveEvent(iNextPosition, pDestinationEventBuffer, _pEventWindowBuffer, _iSizeOfEvent, _iShift);
		}
	}

	memcpy(pDestinationEventBuffer, _pSourceEvent, _iSizeOfEvent);
	pDestinationEvent->i_Type = GpuEvent::EXPIRED;
}

__global__
void SetWindowState(
		char               * _pInputEventBuffer,     // original input events buffer
		int                  _iNumberOfEvents,       // Number of events in input buffer (matched + not matched)
		char               * _pEventWindowBuffer,    // Event window buffer
		int                  _iWindowLength,         // Length of current events window
		int                  _iRemainingCount,       // Remaining free slots in Window buffer
		int                  _iMaxEventCount,        // used for setting results array
		int                  _iSizeOfEvent,          // Size of an event
		int                  _iEventsPerBlock        // number of events allocated per block
)
{
	// avoid out of bound threads
	if(threadIdx.x >= _iEventsPerBlock || threadIdx.y > 0 || blockIdx.y > 0)
		return;

	if((blockIdx.x == _iNumberOfEvents / _iEventsPerBlock) && // last thread block
			(threadIdx.x >= _iNumberOfEvents % _iEventsPerBlock)) // extra threads
	{
		return;
	}

	// get assigned event
	int iEventIdx = (blockIdx.x * _iEventsPerBlock) + threadIdx.x;

	// get in event starting position
	char * pInEvent = _pInputEventBuffer + (_iSizeOfEvent * iEventIdx);

	if(_iNumberOfEvents < _iWindowLength)
	{
		int iWindowPositionShift = _iWindowLength - _iNumberOfEvents;

		if(_iRemainingCount < _iNumberOfEvents)
		{
			int iExitEventCount = _iNumberOfEvents - _iRemainingCount;

			//TODO: make this non recursive
			MoveEvent((iEventIdx + iWindowPositionShift), pInEvent, _pEventWindowBuffer, _iSizeOfEvent, iExitEventCount);

		}
		else
		{
			// just copy event to window
			iWindowPositionShift -= (_iRemainingCount - _iNumberOfEvents);

			char * pWindowEventBuffer = _pEventWindowBuffer + (_iSizeOfEvent * (iEventIdx + iWindowPositionShift));

			memcpy(pWindowEventBuffer, pInEvent, _iSizeOfEvent);
			GpuEvent * pExpiredEvent = (GpuEvent*) pWindowEventBuffer;
			pExpiredEvent->i_Type = GpuEvent::EXPIRED;
		}
	}
	else
	{
		int iWindowPositionShift = _iNumberOfEvents - _iWindowLength;

		if(iEventIdx >= iWindowPositionShift)
		{
			char * pWindowEventBuffer = _pEventWindowBuffer + (_iSizeOfEvent * (iEventIdx - iWindowPositionShift));

			memcpy(pWindowEventBuffer, pInEvent, _iSizeOfEvent);
			GpuEvent * pExpiredEvent = (GpuEvent*) pWindowEventBuffer;
			pExpiredEvent->i_Type = GpuEvent::EXPIRED;
		}
	}
}

__global__
void SetWindowState(
		char               * _pInputEventBuffer,     // original input events buffer
		int                * _pFilterdEventsIndexes, // Matched event indexes from filter kernel
		int                  _iNumberOfEvents,       // Number of events in input buffer (matched + not matched)
		char               * _pEventWindowBuffer,    // Event window buffer
		int                  _iWindowLength,         // Length of current events window
		int                  _iRemainingCount,       // Remaining free slots in Window buffer
		int                  _iMaxEventCount,        // used for setting results array
		int                  _iSizeOfEvent,          // Size of an event
		int                  _iEventsPerBlock        // number of events allocated per block
)
{
	// avoid out of bound threads
	if(threadIdx.x >= _iEventsPerBlock || threadIdx.y > 0 || blockIdx.y > 0)
		return;

	if((blockIdx.x == _iNumberOfEvents / _iEventsPerBlock) && // last thread block
			(threadIdx.x >= _iNumberOfEvents % _iEventsPerBlock)) // extra threads
	{
		return;
	}

	// get assigned event
	int iEventIdx = (blockIdx.x * _iEventsPerBlock) + threadIdx.x;

	// check if event in my index is a matched event
	if(_pFilterdEventsIndexes[iEventIdx] < 0)
	{
		return; // not matched
	}

	// get in event starting position
	char * pInEvent = _pInputEventBuffer + (_iSizeOfEvent * iEventIdx);

	if(_iNumberOfEvents < _iWindowLength)
	{
		int iWindowPositionShift = _iWindowLength - _iNumberOfEvents;

		if(_iRemainingCount < _iNumberOfEvents)
		{
			int iExitEventCount = _iNumberOfEvents - _iRemainingCount;

			//TODO: make this non recursive
			MoveEvent((iEventIdx + iWindowPositionShift), pInEvent, _pEventWindowBuffer, _iSizeOfEvent, iExitEventCount);

		}
		else
		{
			// just copy event to window
			iWindowPositionShift -= (_iRemainingCount - _iNumberOfEvents);

			char * pWindowEventBuffer = _pEventWindowBuffer + (_iSizeOfEvent * (iEventIdx + iWindowPositionShift));

			memcpy(pWindowEventBuffer, pInEvent, _iSizeOfEvent);
			GpuEvent * pExpiredEvent = (GpuEvent*) pWindowEventBuffer;
			pExpiredEvent->i_Type = GpuEvent::EXPIRED;
		}
	}
	else
	{
		int iWindowPositionShift = _iNumberOfEvents - _iWindowLength;

		if(iEventIdx >= iWindowPositionShift)
		{
			char * pWindowEventBuffer = _pEventWindowBuffer + (_iSizeOfEvent * (iEventIdx - iWindowPositionShift));

			memcpy(pWindowEventBuffer, pInEvent, _iSizeOfEvent);
			GpuEvent * pExpiredEvent = (GpuEvent*) pWindowEventBuffer;
			pExpiredEvent->i_Type = GpuEvent::EXPIRED;
		}
	}
}

// ===============================================================================================================================

GpuLengthSlidingWindowFirstKernel::GpuLengthSlidingWindowFirstKernel(GpuProcessor * _pProc, GpuProcessorContext * _pContext,
		int _iThreadBlockSize, int _iWindowSize, FILE * _fPLog) :
	GpuKernel(_pProc, _pContext->GetDeviceId(), _iThreadBlockSize, _fPLog),
	p_Context(_pContext),
	p_InputEventBuffer(NULL),
	p_ResultEventBuffer(NULL),
	p_WindowEventBuffer(NULL),
	b_DeviceSet(false),
	i_WindowSize(_iWindowSize),
	i_RemainingCount(_iWindowSize)
{

}

GpuLengthSlidingWindowFirstKernel::~GpuLengthSlidingWindowFirstKernel()
{
	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] destroy\n");
	fflush(fp_Log);
}

bool GpuLengthSlidingWindowFirstKernel::Initialize(GpuMetaEvent * _pMetaEvent, int _iInputEventBufferSize)
{
	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] Initialize\n");
	fflush(fp_Log);

	// set input event buffer
	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] InpuEventBufferIndex=%d\n", i_InputBufferIndex);
	fflush(fp_Log);
	p_InputEventBuffer = (GpuStreamEventBuffer*) p_Context->GetEventBuffer(i_InputBufferIndex);
	p_InputEventBuffer->Print();

	// set resulting event buffer and its meta data
	p_ResultEventBuffer = new GpuStreamEventBuffer(p_Context->GetDeviceId(), _pMetaEvent, fp_Log);
	p_ResultEventBuffer->CreateEventBuffer(_iInputEventBufferSize * 2);

	i_ResultEventBufferIndex = p_Context->AddEventBuffer(p_ResultEventBuffer);

	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] ResultEventBuffer created : Index=%d Size=%d bytes\n", i_ResultEventBufferIndex,
			p_ResultEventBuffer->GetEventBufferSizeInBytes());
	fflush(fp_Log);
	p_ResultEventBuffer->Print();

	p_WindowEventBuffer = new GpuStreamEventBuffer(p_Context->GetDeviceId(), _pMetaEvent, fp_Log);
	p_WindowEventBuffer->CreateEventBuffer(i_WindowSize);

	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] Created device window buffer : Length=%d Size=%d bytes\n", i_WindowSize,
			p_WindowEventBuffer->GetEventBufferSizeInBytes());
	fflush(fp_Log);

	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] initialize window buffer data \n");
	fflush(fp_Log);
	p_WindowEventBuffer->Print();

	p_WindowEventBuffer->ResetHostEventBuffer(0);

	char * pHostWindowBuffer = p_WindowEventBuffer->GetHostEventBuffer();
	char * pCurrentEvent;
	for(int i=0; i<i_WindowSize; ++i)
	{
		pCurrentEvent = pHostWindowBuffer + (_pMetaEvent->i_SizeOfEventInBytes * i);
		GpuEvent * pGpuEvent = (GpuEvent*) pCurrentEvent;
		pGpuEvent->i_Type = GpuEvent::NONE;
	}

	p_WindowEventBuffer->CopyToDevice(false);

	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] Initialization complete\n");
	fflush(fp_Log);

	return true;
}

void GpuLengthSlidingWindowFirstKernel::Process(int & _iNumEvents, bool _bLast)
{
	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] Process : EventCount=%d WindowRemainingCount=%d\n", _iNumEvents, i_RemainingCount);
	fflush(fp_Log);

#ifdef GPU_DEBUG
	GpuUtils::PrintByteBuffer(p_InputEventBuffer->GetHostEventBuffer(), _iNumEvents, p_InputEventBuffer->GetHostMetaEvent(),
			"GpuLengthSlidingWindowFirstKernel::In", fp_Log);
#endif

	if(!b_DeviceSet)
	{
		GpuCudaHelper::SelectDevice(i_DeviceId, fp_Log);
		b_DeviceSet = true;
	}

	p_InputEventBuffer->CopyToDevice(true);

#ifdef KERNEL_TIME
	sdkStartTimer(&p_StopWatch);
#endif

	// call entry kernel
	int numBlocksX = ceil((float)_iNumEvents / (float)i_ThreadBlockSize);
	int numBlocksY = 1;
	dim3 numBlocks = dim3(numBlocksX, numBlocksY);
	dim3 numThreads = dim3(i_ThreadBlockSize, 1);

#ifdef GPU_DEBUG
	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] Invoke kernel Blocks(%d,%d) Threads(%d,%d)\n", numBlocksX, numBlocksY, i_ThreadBlockSize, 1);
	fflush(fp_Log);
#endif

	ProcessEventsLengthSlidingWindow<<<numBlocks, numThreads>>>(
			p_InputEventBuffer->GetDeviceEventBuffer(),
			p_InputEventBuffer->GetDeviceMetaEvent(),
			_iNumEvents,
			p_WindowEventBuffer->GetDeviceEventBuffer(),
			i_WindowSize,
			i_RemainingCount,
			p_ResultEventBuffer->GetDeviceEventBuffer(),
			p_InputEventBuffer->GetMaxEventCount(),
			p_InputEventBuffer->GetHostMetaEvent()->i_SizeOfEventInBytes,
			i_ThreadBlockSize
	);

	SetWindowState<<<numBlocks, numThreads>>>(
			p_InputEventBuffer->GetDeviceEventBuffer(),
			_iNumEvents,
			p_WindowEventBuffer->GetDeviceEventBuffer(),
			i_WindowSize,
			i_RemainingCount,
			p_InputEventBuffer->GetMaxEventCount(),
			p_InputEventBuffer->GetHostMetaEvent()->i_SizeOfEventInBytes,
			i_ThreadBlockSize
	);

	if(_bLast)
	{
		p_ResultEventBuffer->CopyToHost(true);
#ifdef GPU_DEBUG
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Results copied \n");
	fflush(fp_Log);
#endif
	}

	CUDA_CHECK_RETURN(hipPeekAtLastError());
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

#ifdef GPU_DEBUG
	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] Kernel complete \n");
	fflush(fp_Log);
#endif

#ifdef GPU_DEBUG
	GpuUtils::PrintByteBuffer(p_ResultEventBuffer->GetHostEventBuffer(), _iNumEvents * 2, p_ResultEventBuffer->GetHostMetaEvent(),
			"GpuLengthSlidingWindowFirstKernel::Out", fp_Log);
#endif


#ifdef KERNEL_TIME
	sdkStopTimer(&p_StopWatch);
	float fElapsed = sdkGetTimerValue(&p_StopWatch);
	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] Stats : Elapsed=%f ms\n", fElapsed);
	fflush(fp_Log);
	lst_ElapsedTimes.push_back(fElapsed);
	sdkResetTimer(&p_StopWatch);
#endif

	if(_iNumEvents > i_RemainingCount)
	{
		i_RemainingCount = 0;
	}
	else
	{
		i_RemainingCount -= _iNumEvents;
	}
}

char * GpuLengthSlidingWindowFirstKernel::GetResultEventBuffer()
{
	return p_ResultEventBuffer->GetHostEventBuffer();
}

int GpuLengthSlidingWindowFirstKernel::GetResultEventBufferSize()
{
	return p_ResultEventBuffer->GetEventBufferSizeInBytes();
}

// ===============================================================================================================================

GpuLengthSlidingWindowFilterKernel::GpuLengthSlidingWindowFilterKernel(GpuProcessor * _pProc, GpuProcessorContext * _pContext,
		int _iThreadBlockSize, int _iWindowSize, FILE * _fPLog) :
	GpuKernel(_pProc, _pContext->GetDeviceId(), _iThreadBlockSize, _fPLog),
	p_Context(_pContext),
	p_InputEventBuffer(NULL),
	p_ResultEventBuffer(NULL),
	p_WindowEventBuffer(NULL),
	b_DeviceSet(false),
	i_WindowSize(_iWindowSize),
	i_RemainingCount(_iWindowSize)
{

}

GpuLengthSlidingWindowFilterKernel::~GpuLengthSlidingWindowFilterKernel()
{
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] destroy\n");
	fflush(fp_Log);
}

bool GpuLengthSlidingWindowFilterKernel::Initialize(GpuMetaEvent * _pMetaEvent, int _iInputEventBufferSize)
{
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Initialize\n");
	fflush(fp_Log);

	// set input event buffer
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] InpuEventBufferIndex=%d\n", i_InputBufferIndex);
	fflush(fp_Log);
	p_InputEventBuffer = (GpuStreamEventBuffer*) p_Context->GetEventBuffer(i_InputBufferIndex);
	p_InputEventBuffer->Print();

	// set resulting event buffer and its meta data
	p_ResultEventBuffer = new GpuStreamEventBuffer(p_Context->GetDeviceId(), _pMetaEvent, fp_Log);
	p_ResultEventBuffer->CreateEventBuffer(_iInputEventBufferSize * 2);

	i_ResultEventBufferIndex = p_Context->AddEventBuffer(p_ResultEventBuffer);

	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] ResultEventBuffer created : Index=%d Size=%d bytes\n", i_ResultEventBufferIndex,
			p_ResultEventBuffer->GetEventBufferSizeInBytes());
	fflush(fp_Log);
	p_ResultEventBuffer->Print();

	p_WindowEventBuffer = new GpuStreamEventBuffer(p_Context->GetDeviceId(), _pMetaEvent, fp_Log);
	p_WindowEventBuffer->CreateEventBuffer(i_WindowSize);

	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Created device window buffer : Length=%d Size=%d bytes\n", i_WindowSize,
			p_WindowEventBuffer->GetEventBufferSizeInBytes());
	fflush(fp_Log);
	p_WindowEventBuffer->Print();

	p_WindowEventBuffer->ResetHostEventBuffer(0);

	char * pHostWindowBuffer = p_WindowEventBuffer->GetHostEventBuffer();
	char * pCurrentEvent;
	for(int i=0; i<i_WindowSize; ++i)
	{
		pCurrentEvent = pHostWindowBuffer + (_pMetaEvent->i_SizeOfEventInBytes * i);
		GpuEvent * pGpuEvent = (GpuEvent*) pCurrentEvent;
		pGpuEvent->i_Type = GpuEvent::NONE;
	}

	p_WindowEventBuffer->CopyToDevice(false);

	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Initialization complete\n");
	fflush(fp_Log);

	return true;
}

void GpuLengthSlidingWindowFilterKernel::Process(int & _iNumEvents, bool _bLast)
{
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Process : EventCount=%d\n", _iNumEvents);
	fflush(fp_Log);

#ifdef GPU_DEBUG
	GpuUtils::PrintByteBuffer(p_InputEventBuffer->GetHostEventBuffer(), _iNumEvents, p_InputEventBuffer->GetHostMetaEvent(),
			"GpuLengthSlidingWindowFilterKernel::In", fp_Log);
#endif

	if(!b_DeviceSet) // TODO: check if this works in every conditions. How Java thread pool works with disrupter?
	{
		GpuCudaHelper::SelectDevice(i_DeviceId, fp_Log);
		b_DeviceSet = true;
	}

#ifdef KERNEL_TIME
	sdkStartTimer(&p_StopWatch);
#endif

	// call entry kernel
	int numBlocksX = ceil((float)_iNumEvents / (float)i_ThreadBlockSize);
	int numBlocksY = 1;
	dim3 numBlocks = dim3(numBlocksX, numBlocksY);
	dim3 numThreads = dim3(i_ThreadBlockSize, 1);

#ifdef GPU_DEBUG
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Invoke kernel Blocks(%d,%d) Threads(%d,%d)\n", numBlocksX, numBlocksY, i_ThreadBlockSize, 1);
	fflush(fp_Log);
#endif

	ProcessEventsLengthSlidingWindow<<<numBlocks, numThreads>>>(
			p_InputEventBuffer->GetDeviceEventBuffer(),
			p_InputEventBuffer->GetDeviceMetaEvent(),
			_iNumEvents,
			p_WindowEventBuffer->GetDeviceEventBuffer(),
			i_WindowSize,
			i_RemainingCount,
			p_ResultEventBuffer->GetDeviceEventBuffer(),
			p_InputEventBuffer->GetMaxEventCount(),
			p_InputEventBuffer->GetHostMetaEvent()->i_SizeOfEventInBytes,
			i_ThreadBlockSize
	);

	SetWindowState<<<numBlocks, numThreads>>>(
			p_InputEventBuffer->GetDeviceEventBuffer(),
			_iNumEvents,
			p_WindowEventBuffer->GetDeviceEventBuffer(),
			i_WindowSize,
			i_RemainingCount,
			p_InputEventBuffer->GetMaxEventCount(),
			p_InputEventBuffer->GetHostMetaEvent()->i_SizeOfEventInBytes,
			i_ThreadBlockSize
	);

#ifdef GPU_DEBUG
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Kernel complete \n");
	fflush(fp_Log);
#endif

	if(_bLast)
	{
		p_ResultEventBuffer->CopyToHost(true);

#ifdef GPU_DEBUG
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Results copied \n");
	fflush(fp_Log);
#endif
	}

	CUDA_CHECK_RETURN(hipPeekAtLastError());
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

#ifdef GPU_DEBUG
	GpuUtils::PrintByteBuffer(p_ResultEventBuffer->GetHostEventBuffer(), _iNumEvents * 2, p_ResultEventBuffer->GetHostMetaEvent(),
			"GpuLengthSlidingWindowFilterKernel::Out", fp_Log);
#endif


#ifdef KERNEL_TIME
	sdkStopTimer(&p_StopWatch);
	float fElapsed = sdkGetTimerValue(&p_StopWatch);
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Stats : Elapsed=%f ms\n", fElapsed);
	fflush(fp_Log);
	lst_ElapsedTimes.push_back(fElapsed);
	sdkResetTimer(&p_StopWatch);
#endif

	if(_iNumEvents > i_RemainingCount)
	{
		i_RemainingCount = 0;
	}
	else
	{
		i_RemainingCount -= _iNumEvents;
	}
}

char * GpuLengthSlidingWindowFilterKernel::GetResultEventBuffer()
{
	return p_ResultEventBuffer->GetHostEventBuffer();
}

int GpuLengthSlidingWindowFilterKernel::GetResultEventBufferSize()
{
	return p_ResultEventBuffer->GetEventBufferSizeInBytes();
}

}

#endif
