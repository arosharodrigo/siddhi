#include "hip/hip_runtime.h"
#ifndef _GPU_LENGTH_SLIDING_WINDOW_KERNEL_CU__
#define _GPU_LENGTH_SLIDING_WINDOW_KERNEL_CU__

#include <stdio.h>
#include <stdlib.h>

#include "GpuMetaEvent.h"
#include "GpuProcessor.h"
#include "GpuProcessorContext.h"
#include "GpuStreamEventBuffer.h"
#include "GpuIntBuffer.h"
#include "GpuKernelDataTypes.h"
#include "GpuLengthSlidingWindowKernel.h"
#include "GpuCudaHelper.h"
#include "GpuUtils.h"

namespace SiddhiGpu
{

__global__
void ProcessEventsLengthSlidingWindow(
		char               * _pInputEventBuffer,     // original input events buffer
		GpuKernelMetaEvent * _pMetaEvent,            // Meta event for original input events
		int                  _iNumberOfEvents,       // Number of events in input buffer (matched + not matched)
		char               * _pEventWindowBuffer,    // Event window buffer
		int                  _iWindowLength,         // Length of current events window
		int                  _iRemainingCount,       // Remaining free slots in Window buffer
		char               * _pResultsBuffer,        // Resulting events buffer
		int                  _iMaxEventCount,        // used for setting results array
		int                  _iSizeOfEvent,          // Size of an event
		int                  _iEventsPerBlock        // number of events allocated per block
)
{
	// avoid out of bound threads
	if(threadIdx.x >= _iEventsPerBlock || threadIdx.y > 0 || blockIdx.y > 0)
		return;

	if((blockIdx.x == _iNumberOfEvents / _iEventsPerBlock) && // last thread block
			(threadIdx.x >= _iNumberOfEvents % _iEventsPerBlock)) // extra threads
	{
		return;
	}

	// get assigned event
	int iEventIdx = (blockIdx.x * _iEventsPerBlock) + threadIdx.x;

	// get in event starting position
	char * pInEventBuffer = _pInputEventBuffer + (_iSizeOfEvent * iEventIdx);

	// output to results buffer [expired event, in event]
	char * pResultsExpiredEventBuffer = _pResultsBuffer + (_iSizeOfEvent * iEventIdx * 2);
	char * pResultsInEventBuffer = pResultsExpiredEventBuffer + _iSizeOfEvent;

	GpuEvent * pExpiredEvent = (GpuEvent *)pResultsExpiredEventBuffer;
	// calculate in/expired event pair for this event

	memcpy(pResultsInEventBuffer, pInEventBuffer, _iSizeOfEvent);

	if(iEventIdx >= _iRemainingCount)
	{
		if(iEventIdx < _iWindowLength)
		{
			// in window buffer
			char * pExpiredOutEventInWindowBuffer = _pEventWindowBuffer + (_iSizeOfEvent * (iEventIdx - _iRemainingCount));

			GpuEvent * pWindowEvent = (GpuEvent*) pExpiredOutEventInWindowBuffer;
			if(pWindowEvent->i_Type != GpuEvent::NONE) // if window event is filled
			{
				memcpy(pResultsExpiredEventBuffer, pExpiredOutEventInWindowBuffer, _iSizeOfEvent);
				pExpiredEvent->i_Type = GpuEvent::EXPIRED;
			}
			else
			{
				memset(pResultsExpiredEventBuffer, 0, _iSizeOfEvent);
				pExpiredEvent->i_Type = GpuEvent::NONE;
			}
		}
		else
		{
			// in input event buffer
			char * pExpiredOutEventInInputBuffer = _pInputEventBuffer + (_iSizeOfEvent * (iEventIdx - _iWindowLength));

			memcpy(pResultsExpiredEventBuffer, pExpiredOutEventInInputBuffer, _iSizeOfEvent);
			pExpiredEvent->i_Type = GpuEvent::EXPIRED;
		}
	}
	else
	{
		// [NULL,inEvent]
		memset(pResultsExpiredEventBuffer, 0, _iSizeOfEvent);
		pExpiredEvent->i_Type = GpuEvent::NONE;

	}

}

 // not used
//__global__
//void ProcessEventsLengthSlidingWindowFilter(
//		char               * _pInputEventBuffer,     // original input events buffer
//		GpuKernelMetaEvent * _pMetaEvent,            // Meta event for original input events
//		int                  _iNumberOfEvents,       // Number of events in input buffer (matched + not matched)
//		int                * _pFilterdEventsIndexes, // Matched event indexes from filter kernel
//		char               * _pEventWindowBuffer,    // Event window buffer
//		int                  _iWindowLength,         // Length of current events window
//		int                  _iRemainingCount,       // Remaining free slots in Window buffer
//		char               * _pResultsBuffer,        // Resulting events buffer
//		int                  _iMaxEventCount,        // used for setting results array
//		int                  _iSizeOfEvent,          // Size of an event
//		int                  _iEventsPerBlock        // number of events allocated per block
//)
//{
//	// avoid out of bound threads
//	if(threadIdx.x >= _iEventsPerBlock || threadIdx.y > 0 || blockIdx.y > 0)
//		return;
//
//	if((blockIdx.x == _iNumberOfEvents / _iEventsPerBlock) && // last thread block
//			(threadIdx.x >= _iNumberOfEvents % _iEventsPerBlock)) // extra threads
//	{
//		return;
//	}
//
//	// get assigned event
//	int iEventIdx = (blockIdx.x * _iEventsPerBlock) + threadIdx.x;
//
//	// get in event starting position
//	char * pInEventBuffer = _pInputEventBuffer + (_iSizeOfEvent * iEventIdx);
//
//	// output to results buffer [expired event, in event]
//	char * pResultsExpiredEventBuffer = _pResultsBuffer + (_iSizeOfEvent * iEventIdx * 2);
//	char * pResultsInEventBuffer = pResultsExpiredEventBuffer + _iSizeOfEvent;
//
//	GpuEvent * pExpiredEvent = (GpuEvent *)pResultsExpiredEventBuffer;
//
//	// check if event in my index is a matched event
//	if(_pFilterdEventsIndexes[iEventIdx] < 0)
//	{
//		memset(pResultsExpiredEventBuffer, 0, _iSizeOfEvent);
//		pExpiredEvent->i_Type = GpuEvent::NONE;
//
//		memset(pResultsInEventBuffer, 0, _iSizeOfEvent);
//		GpuEvent * pResultsInEvent = (GpuEvent *)pResultsInEventBuffer;
//		pResultsInEvent->i_Type = GpuEvent::NONE;
//
//		return; // not matched
//	}
//
//	// calculate in/expired event pair for this event
//
//
//	memcpy(pResultsInEventBuffer, pInEventBuffer, _iSizeOfEvent);
//
//	if(iEventIdx >= _iRemainingCount)
//	{
//		if(iEventIdx < _iWindowLength)
//		{
//			// in window buffer
//			char * pExpiredOutEventInWindowBuffer = _pEventWindowBuffer + (_iSizeOfEvent * (iEventIdx - _iRemainingCount));
//
//			GpuEvent * pWindowEvent = (GpuEvent*) pExpiredOutEventInWindowBuffer;
//			if(pWindowEvent->i_Type != GpuEvent::NONE) // if window event is filled
//			{
//				memcpy(pResultsExpiredEventBuffer, pExpiredOutEventInWindowBuffer, _iSizeOfEvent);
//				pExpiredEvent->i_Type = GpuEvent::EXPIRED;
//			}
//			else
//			{
//				memset(pResultsExpiredEventBuffer, 0, _iSizeOfEvent);
//				pExpiredEvent->i_Type = GpuEvent::NONE;
//			}
//		}
//		else
//		{
//			// in input event buffer
//			char * pExpiredOutEventInInputBuffer = _pInputEventBuffer + (_iSizeOfEvent * (iEventIdx - _iWindowLength));
//
//			memcpy(pResultsExpiredEventBuffer, pExpiredOutEventInInputBuffer, _iSizeOfEvent);
//			pExpiredEvent->i_Type = GpuEvent::EXPIRED;
//		}
//	}
//	else
//	{
//		// [NULL,inEvent]
//		memset(pResultsExpiredEventBuffer, 0, _iSizeOfEvent);
//		pExpiredEvent->i_Type = GpuEvent::NONE;
//
//	}
//
//}

__global__
void SetWindowState(
		char               * _pInputEventBuffer,     // original input events buffer
		int                  _iNumberOfEvents,       // Number of events in input buffer (matched + not matched)
		char               * _pEventWindowBuffer,    // Event window buffer
		int                  _iWindowLength,         // Length of current events window
		int                  _iRemainingCount,       // Remaining free slots in Window buffer
		int                  _iMaxEventCount,        // used for setting results array
		int                  _iSizeOfEvent,          // Size of an event
		int                  _iEventsPerBlock        // number of events allocated per block
)
{
	// avoid out of bound threads
	if(threadIdx.x >= _iEventsPerBlock || threadIdx.y > 0 || blockIdx.y > 0)
		return;

	if((blockIdx.x == _iNumberOfEvents / _iEventsPerBlock) && // last thread block
			(threadIdx.x >= _iNumberOfEvents % _iEventsPerBlock)) // extra threads
	{
		return;
	}

	// get assigned event
	int iEventIdx = (blockIdx.x * _iEventsPerBlock) + threadIdx.x;

	// get in event starting position
	char * pInEventBuffer = _pInputEventBuffer + (_iSizeOfEvent * iEventIdx);

	if(_iNumberOfEvents < _iWindowLength)
	{
		int iWindowPositionShift = _iWindowLength - _iNumberOfEvents;

		if(_iRemainingCount < _iNumberOfEvents)
		{
			int iExitEventCount = _iNumberOfEvents - _iRemainingCount;

			// calculate start and end window buffer positions
			int iStart = iEventIdx + iWindowPositionShift;
			int iEnd = iStart;
			int iPrevToEnd = iEnd;
			while(iEnd >= 0)
			{
				char * pDestinationEventBuffer = _pEventWindowBuffer + (_iSizeOfEvent * iEnd);
				GpuEvent * pDestinationEvent = (GpuEvent*) pDestinationEventBuffer;

				if(pDestinationEvent->i_Type != GpuEvent::NONE) // there is an event in destination position
				{
					iPrevToEnd = iEnd;
					iEnd -= iExitEventCount;
				}
				else
				{
					break;
				}

			}

			iEnd = (iEnd < 0 ? iPrevToEnd : iEnd);

			// work back from end while copying events
			while(iEnd < iStart)
			{
				char * pDestinationEventBuffer = _pEventWindowBuffer + (_iSizeOfEvent * iEnd);
				GpuEvent * pDestinationEvent = (GpuEvent*) pDestinationEventBuffer;

				char * pSourceEventBuffer = _pEventWindowBuffer + (_iSizeOfEvent * (iEnd + iExitEventCount));

				memcpy(pDestinationEventBuffer, pSourceEventBuffer, _iSizeOfEvent);
				pDestinationEvent->i_Type = GpuEvent::EXPIRED;

				iEnd += iExitEventCount;
			}

			// iEnd == iStart
			if(iStart >= 0)
			{
				char * pDestinationEventBuffer = _pEventWindowBuffer + (_iSizeOfEvent * iStart);
				GpuEvent * pDestinationEvent = (GpuEvent*) pDestinationEventBuffer;
				memcpy(pDestinationEventBuffer, pInEventBuffer, _iSizeOfEvent);
				pDestinationEvent->i_Type = GpuEvent::EXPIRED;
			}
		}
		else
		{
			// just copy event to window
			iWindowPositionShift -= (_iRemainingCount - _iNumberOfEvents);

			char * pWindowEventBuffer = _pEventWindowBuffer + (_iSizeOfEvent * (iEventIdx + iWindowPositionShift));

			memcpy(pWindowEventBuffer, pInEventBuffer, _iSizeOfEvent);
			GpuEvent * pExpiredEvent = (GpuEvent*) pWindowEventBuffer;
			pExpiredEvent->i_Type = GpuEvent::EXPIRED;
		}
	}
	else
	{
		int iWindowPositionShift = _iNumberOfEvents - _iWindowLength;

		if(iEventIdx >= iWindowPositionShift)
		{
			char * pWindowEventBuffer = _pEventWindowBuffer + (_iSizeOfEvent * (iEventIdx - iWindowPositionShift));

			memcpy(pWindowEventBuffer, pInEventBuffer, _iSizeOfEvent);
			GpuEvent * pExpiredEvent = (GpuEvent*) pWindowEventBuffer;
			pExpiredEvent->i_Type = GpuEvent::EXPIRED;
		}
	}
}

// ===============================================================================================================================

GpuLengthSlidingWindowFirstKernel::GpuLengthSlidingWindowFirstKernel(GpuProcessor * _pProc, GpuProcessorContext * _pContext,
		int _iThreadBlockSize, int _iWindowSize, FILE * _fPLog) :
	GpuKernel(_pProc, _pContext->GetDeviceId(), _iThreadBlockSize, _fPLog),
	p_Context(_pContext),
	p_InputEventBuffer(NULL),
	p_ResultEventBuffer(NULL),
	p_WindowEventBuffer(NULL),
	b_DeviceSet(false),
	i_WindowSize(_iWindowSize),
	i_RemainingCount(_iWindowSize)
{

}

GpuLengthSlidingWindowFirstKernel::~GpuLengthSlidingWindowFirstKernel()
{
	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] destroy\n");
	fflush(fp_Log);

	if(p_DeviceOutputAttributeMapping)
	{
		CUDA_CHECK_RETURN(hipFree(p_DeviceOutputAttributeMapping));
		p_DeviceOutputAttributeMapping = NULL;
	}
}

bool GpuLengthSlidingWindowFirstKernel::Initialize(int _iStreamIndex, GpuMetaEvent * _pMetaEvent, int _iInputEventBufferSize)
{
	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] Initialize : StreamIndex=%d \n", _iStreamIndex);
	fflush(fp_Log);

	// set input event buffer
	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] InpuEventBufferIndex=%d\n", i_InputBufferIndex);
	fflush(fp_Log);
	p_InputEventBuffer = (GpuStreamEventBuffer*) p_Context->GetEventBuffer(i_InputBufferIndex);
	p_InputEventBuffer->Print();

	// set resulting event buffer and its meta data
	p_ResultEventBuffer = new GpuStreamEventBuffer("WindowResultEventBuffer", p_Context->GetDeviceId(), _pMetaEvent, fp_Log);
	p_ResultEventBuffer->CreateEventBuffer(_iInputEventBufferSize * 2);

	i_ResultEventBufferIndex = p_Context->AddEventBuffer(p_ResultEventBuffer);

	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] ResultEventBuffer created : Index=%d Size=%d bytes\n", i_ResultEventBufferIndex,
			p_ResultEventBuffer->GetEventBufferSizeInBytes());
	fflush(fp_Log);
	p_ResultEventBuffer->Print();

	p_WindowEventBuffer = new GpuStreamEventBuffer("WindowEventBuffer", p_Context->GetDeviceId(), _pMetaEvent, fp_Log);
	p_WindowEventBuffer->CreateEventBuffer(i_WindowSize);

	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] Created device window buffer : Length=%d Size=%d bytes\n", i_WindowSize,
			p_WindowEventBuffer->GetEventBufferSizeInBytes());
	fflush(fp_Log);

	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] initialize window buffer data \n");
	fflush(fp_Log);
	p_WindowEventBuffer->Print();

	p_WindowEventBuffer->ResetHostEventBuffer(0);

	char * pHostWindowBuffer = p_WindowEventBuffer->GetHostEventBuffer();
	char * pCurrentEvent;
	for(int i=0; i<i_WindowSize; ++i)
	{
		pCurrentEvent = pHostWindowBuffer + (_pMetaEvent->i_SizeOfEventInBytes * i);
		GpuEvent * pGpuEvent = (GpuEvent*) pCurrentEvent;
		pGpuEvent->i_Type = GpuEvent::NONE;
	}

	p_WindowEventBuffer->CopyToDevice(false);

	// copy Output mappings
	if(p_HostOutputAttributeMapping)
	{
		fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] Copying AttributeMappings to device \n");
		fflush(fp_Log);

		fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] AttributeMapCount : %d \n", p_HostOutputAttributeMapping->i_MappingCount);
		for(int c=0; c<p_HostOutputAttributeMapping->i_MappingCount; ++c)
		{
			fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] Map : Form [Stream=%d, Attrib=%d] To [Attrib=%d] \n",
					p_HostOutputAttributeMapping->p_Mappings[c].from[AttributeMapping::STREAM_INDEX],
					p_HostOutputAttributeMapping->p_Mappings[c].from[AttributeMapping::ATTRIBUTE_INDEX],
					p_HostOutputAttributeMapping->p_Mappings[c].to);

		}

		CUDA_CHECK_RETURN(hipMalloc(
				(void**) &p_DeviceOutputAttributeMapping,
				sizeof(AttributeMappings)));

		AttributeMappings * pHostMappings = (AttributeMappings*) malloc(sizeof(AttributeMappings));
		pHostMappings->i_MappingCount = p_HostOutputAttributeMapping->i_MappingCount;
		pHostMappings->p_Mappings = NULL;

		CUDA_CHECK_RETURN(hipMalloc(
				(void**) &pHostMappings->p_Mappings,
				sizeof(AttributeMapping) * p_HostOutputAttributeMapping->i_MappingCount));

		CUDA_CHECK_RETURN(hipMemcpy(
				pHostMappings->p_Mappings,
				p_HostOutputAttributeMapping->p_Mappings,
				sizeof(AttributeMapping) * p_HostOutputAttributeMapping->i_MappingCount,
				hipMemcpyHostToDevice));

		CUDA_CHECK_RETURN(hipMemcpy(
				p_DeviceOutputAttributeMapping,
				pHostMappings,
				sizeof(AttributeMappings),
				hipMemcpyHostToDevice));

		CUDA_CHECK_RETURN(hipPeekAtLastError());
		CUDA_CHECK_RETURN(hipDeviceSynchronize());

		free(pHostMappings);
		pHostMappings = NULL;
	}

	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] Initialization complete\n");
	fflush(fp_Log);

	return true;
}

void GpuLengthSlidingWindowFirstKernel::Process(int _iStreamIndex, int & _iNumEvents)
{
#if GPU_DEBUG >= GPU_DEBUG_LEVEL_TRACE
	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] Process : EventCount=%d WindowRemainingCount=%d\n", _iNumEvents, i_RemainingCount);
	fflush(fp_Log);

	GpuUtils::PrintByteBuffer(p_InputEventBuffer->GetHostEventBuffer(), _iNumEvents, p_InputEventBuffer->GetHostMetaEvent(),
			"GpuLengthSlidingWindowFirstKernel::In", fp_Log);
#endif

	if(!b_DeviceSet)
	{
		GpuCudaHelper::SelectDevice(i_DeviceId, "GpuLengthSlidingWindowFirstKernel", fp_Log);
		b_DeviceSet = true;
	}

	p_InputEventBuffer->CopyToDevice(true);

#ifdef KERNEL_TIME
	sdkStartTimer(&p_StopWatch);
#endif

	// call entry kernel
	int numBlocksX = ceil((float)_iNumEvents / (float)i_ThreadBlockSize);
	int numBlocksY = 1;
	dim3 numBlocks = dim3(numBlocksX, numBlocksY);
	dim3 numThreads = dim3(i_ThreadBlockSize, 1);

#if GPU_DEBUG >= GPU_DEBUG_LEVEL_INFO
	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] Invoke kernel Blocks(%d,%d) Threads(%d,%d)\n", numBlocksX, numBlocksY, i_ThreadBlockSize, 1);
	fflush(fp_Log);
#endif

	ProcessEventsLengthSlidingWindow<<<numBlocks, numThreads>>>(
			p_InputEventBuffer->GetDeviceEventBuffer(),
			p_InputEventBuffer->GetDeviceMetaEvent(),
			_iNumEvents,
			p_WindowEventBuffer->GetDeviceEventBuffer(),
			i_WindowSize,
			i_RemainingCount,
			p_ResultEventBuffer->GetDeviceEventBuffer(),
			p_InputEventBuffer->GetMaxEventCount(),
			p_InputEventBuffer->GetHostMetaEvent()->i_SizeOfEventInBytes,
			i_ThreadBlockSize
	);

	if(b_LastKernel)
	{
		p_ResultEventBuffer->CopyToHost(true);
#if GPU_DEBUG >= GPU_DEBUG_LEVEL_DEBUG
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Results copied \n");
	fflush(fp_Log);
#endif
	}

	SetWindowState<<<numBlocks, numThreads>>>(
			p_InputEventBuffer->GetDeviceEventBuffer(),
			_iNumEvents,
			p_WindowEventBuffer->GetDeviceEventBuffer(),
			i_WindowSize,
			i_RemainingCount,
			p_InputEventBuffer->GetMaxEventCount(),
			p_InputEventBuffer->GetHostMetaEvent()->i_SizeOfEventInBytes,
			i_ThreadBlockSize
	);

	CUDA_CHECK_RETURN(hipPeekAtLastError());
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

#if GPU_DEBUG >= GPU_DEBUG_LEVEL_TRACE
	if(b_LastKernel)
	{
		GpuUtils::PrintByteBuffer(p_ResultEventBuffer->GetHostEventBuffer(), _iNumEvents * 2, p_ResultEventBuffer->GetHostMetaEvent(),
				"GpuLengthSlidingWindowFirstKernel::Out", fp_Log);
	}
	fflush(fp_Log);
#endif

#if GPU_DEBUG >= GPU_DEBUG_LEVEL_INFO
	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] Kernel complete \n");
	fflush(fp_Log);
#endif


#ifdef KERNEL_TIME
	sdkStopTimer(&p_StopWatch);
	float fElapsed = sdkGetTimerValue(&p_StopWatch);
	fprintf(fp_Log, "[GpuLengthSlidingWindowFirstKernel] Stats : Elapsed=%f ms\n", fElapsed);
	fflush(fp_Log);
	lst_ElapsedTimes.push_back(fElapsed);
	sdkResetTimer(&p_StopWatch);
#endif

	if(_iNumEvents > i_RemainingCount)
	{
		i_RemainingCount = 0;
	}
	else
	{
		i_RemainingCount -= _iNumEvents;
	}
}

char * GpuLengthSlidingWindowFirstKernel::GetResultEventBuffer()
{
	return p_ResultEventBuffer->GetHostEventBuffer();
}

int GpuLengthSlidingWindowFirstKernel::GetResultEventBufferSize()
{
	return p_ResultEventBuffer->GetEventBufferSizeInBytes();
}

// ===============================================================================================================================

GpuLengthSlidingWindowFilterKernel::GpuLengthSlidingWindowFilterKernel(GpuProcessor * _pProc, GpuProcessorContext * _pContext,
		int _iThreadBlockSize, int _iWindowSize, FILE * _fPLog) :
	GpuKernel(_pProc, _pContext->GetDeviceId(), _iThreadBlockSize, _fPLog),
	p_Context(_pContext),
	p_InputEventBuffer(NULL),
	p_ResultEventBuffer(NULL),
	p_WindowEventBuffer(NULL),
	b_DeviceSet(false),
	i_WindowSize(_iWindowSize),
	i_RemainingCount(_iWindowSize)
{

}

GpuLengthSlidingWindowFilterKernel::~GpuLengthSlidingWindowFilterKernel()
{
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] destroy\n");
	fflush(fp_Log);

	if(p_DeviceOutputAttributeMapping)
	{
		CUDA_CHECK_RETURN(hipFree(p_DeviceOutputAttributeMapping));
		p_DeviceOutputAttributeMapping = NULL;
	}
}

bool GpuLengthSlidingWindowFilterKernel::Initialize(int _iStreamIndex, GpuMetaEvent * _pMetaEvent, int _iInputEventBufferSize)
{
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Initialize : StreamIndex=%d \n", _iStreamIndex);
	fflush(fp_Log);

	// set input event buffer
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] InpuEventBufferIndex=%d\n", i_InputBufferIndex);
	fflush(fp_Log);
	p_InputEventBuffer = (GpuStreamEventBuffer*) p_Context->GetEventBuffer(i_InputBufferIndex);
	p_InputEventBuffer->Print();

	// set resulting event buffer and its meta data
	p_ResultEventBuffer = new GpuStreamEventBuffer("WindowResultEventBuffer", p_Context->GetDeviceId(), _pMetaEvent, fp_Log);
	p_ResultEventBuffer->CreateEventBuffer(_iInputEventBufferSize * 2);

	i_ResultEventBufferIndex = p_Context->AddEventBuffer(p_ResultEventBuffer);

	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] ResultEventBuffer created : Index=%d Size=%d bytes\n", i_ResultEventBufferIndex,
			p_ResultEventBuffer->GetEventBufferSizeInBytes());
	fflush(fp_Log);
	p_ResultEventBuffer->Print();

	p_WindowEventBuffer = new GpuStreamEventBuffer("WindowEventBuffer", p_Context->GetDeviceId(), _pMetaEvent, fp_Log);
	p_WindowEventBuffer->CreateEventBuffer(i_WindowSize);

	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Created device window buffer : Length=%d Size=%d bytes\n", i_WindowSize,
			p_WindowEventBuffer->GetEventBufferSizeInBytes());
	fflush(fp_Log);
	p_WindowEventBuffer->Print();

	p_WindowEventBuffer->ResetHostEventBuffer(0);

	char * pHostWindowBuffer = p_WindowEventBuffer->GetHostEventBuffer();
	char * pCurrentEvent;
	for(int i=0; i<i_WindowSize; ++i)
	{
		pCurrentEvent = pHostWindowBuffer + (_pMetaEvent->i_SizeOfEventInBytes * i);
		GpuEvent * pGpuEvent = (GpuEvent*) pCurrentEvent;
		pGpuEvent->i_Type = GpuEvent::NONE;
	}

	p_WindowEventBuffer->CopyToDevice(false);

	// copy Output mappings
	if(p_HostOutputAttributeMapping)
	{
		fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Copying AttributeMappings to device \n");
		fflush(fp_Log);

		fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] AttributeMapCount : %d \n", p_HostOutputAttributeMapping->i_MappingCount);
		for(int c=0; c<p_HostOutputAttributeMapping->i_MappingCount; ++c)
		{
			fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Map : Form [Stream=%d, Attrib=%d] To [Attrib=%d] \n",
					p_HostOutputAttributeMapping->p_Mappings[c].from[AttributeMapping::STREAM_INDEX],
					p_HostOutputAttributeMapping->p_Mappings[c].from[AttributeMapping::ATTRIBUTE_INDEX],
					p_HostOutputAttributeMapping->p_Mappings[c].to);

		}

		CUDA_CHECK_RETURN(hipMalloc(
				(void**) &p_DeviceOutputAttributeMapping,
				sizeof(AttributeMappings)));

		AttributeMappings * pHostMappings = (AttributeMappings*) malloc(sizeof(AttributeMappings));
		pHostMappings->i_MappingCount = p_HostOutputAttributeMapping->i_MappingCount;
		pHostMappings->p_Mappings = NULL;

		CUDA_CHECK_RETURN(hipMalloc(
				(void**) &pHostMappings->p_Mappings,
				sizeof(AttributeMapping) * p_HostOutputAttributeMapping->i_MappingCount));

		CUDA_CHECK_RETURN(hipMemcpy(
				pHostMappings->p_Mappings,
				p_HostOutputAttributeMapping->p_Mappings,
				sizeof(AttributeMapping) * p_HostOutputAttributeMapping->i_MappingCount,
				hipMemcpyHostToDevice));

		CUDA_CHECK_RETURN(hipMemcpy(
				p_DeviceOutputAttributeMapping,
				pHostMappings,
				sizeof(AttributeMappings),
				hipMemcpyHostToDevice));

		CUDA_CHECK_RETURN(hipPeekAtLastError());
		CUDA_CHECK_RETURN(hipDeviceSynchronize());

		free(pHostMappings);
		pHostMappings = NULL;
	}

	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Initialization complete\n");
	fflush(fp_Log);

	return true;
}

void GpuLengthSlidingWindowFilterKernel::Process(int _iStreamIndex, int & _iNumEvents)
{
#if GPU_DEBUG >= GPU_DEBUG_LEVEL_TRACE
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Process : EventCount=%d\n", _iNumEvents);
	fflush(fp_Log);

	p_InputEventBuffer->CopyToHost(true);
	CUDA_CHECK_RETURN(hipPeekAtLastError());
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	GpuUtils::PrintByteBuffer(p_InputEventBuffer->GetHostEventBuffer(), _iNumEvents, p_InputEventBuffer->GetHostMetaEvent(),
			"GpuLengthSlidingWindowFilterKernel::In", fp_Log);
#endif

	if(!b_DeviceSet)
	{
		GpuCudaHelper::SelectDevice(i_DeviceId, "GpuLengthSlidingWindowFilterKernel", fp_Log);
		b_DeviceSet = true;
	}

#ifdef KERNEL_TIME
	sdkStartTimer(&p_StopWatch);
#endif

	// call entry kernel
	int numBlocksX = ceil((float)_iNumEvents / (float)i_ThreadBlockSize);
	int numBlocksY = 1;
	dim3 numBlocks = dim3(numBlocksX, numBlocksY);
	dim3 numThreads = dim3(i_ThreadBlockSize, 1);

#if GPU_DEBUG >= GPU_DEBUG_LEVEL_INFO
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Invoke kernel Blocks(%d,%d) Threads(%d,%d)\n", numBlocksX, numBlocksY, i_ThreadBlockSize, 1);
	fflush(fp_Log);
#endif

	ProcessEventsLengthSlidingWindow<<<numBlocks, numThreads>>>(
			p_InputEventBuffer->GetDeviceEventBuffer(),
			p_InputEventBuffer->GetDeviceMetaEvent(),
			_iNumEvents,
			p_WindowEventBuffer->GetDeviceEventBuffer(),
			i_WindowSize,
			i_RemainingCount,
			p_ResultEventBuffer->GetDeviceEventBuffer(),
			p_InputEventBuffer->GetMaxEventCount(),
			p_InputEventBuffer->GetHostMetaEvent()->i_SizeOfEventInBytes,
			i_ThreadBlockSize
	);

	if(b_LastKernel)
	{
		p_ResultEventBuffer->CopyToHost(true);

#if GPU_DEBUG >= GPU_DEBUG_LEVEL_DEBUG
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Results copied \n");
	fflush(fp_Log);
#endif
	}

	SetWindowState<<<numBlocks, numThreads>>>(
			p_InputEventBuffer->GetDeviceEventBuffer(),
			_iNumEvents,
			p_WindowEventBuffer->GetDeviceEventBuffer(),
			i_WindowSize,
			i_RemainingCount,
			p_InputEventBuffer->GetMaxEventCount(),
			p_InputEventBuffer->GetHostMetaEvent()->i_SizeOfEventInBytes,
			i_ThreadBlockSize
	);

	CUDA_CHECK_RETURN(hipPeekAtLastError());
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

#if GPU_DEBUG >= GPU_DEBUG_LEVEL_TRACE
	if(b_LastKernel)
	{
		GpuUtils::PrintByteBuffer(p_ResultEventBuffer->GetHostEventBuffer(), _iNumEvents * 2, p_ResultEventBuffer->GetHostMetaEvent(),
				"GpuLengthSlidingWindowFilterKernel::Out", fp_Log);
	}
	fflush(fp_Log);
#endif

#if GPU_DEBUG >= GPU_DEBUG_LEVEL_INFO
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Kernel complete \n");
	fflush(fp_Log);
#endif


#ifdef KERNEL_TIME
	sdkStopTimer(&p_StopWatch);
	float fElapsed = sdkGetTimerValue(&p_StopWatch);
	fprintf(fp_Log, "[GpuLengthSlidingWindowFilterKernel] Stats : Elapsed=%f ms\n", fElapsed);
	fflush(fp_Log);
	lst_ElapsedTimes.push_back(fElapsed);
	sdkResetTimer(&p_StopWatch);
#endif

	if(_iNumEvents > i_RemainingCount)
	{
		i_RemainingCount = 0;
	}
	else
	{
		i_RemainingCount -= _iNumEvents;
	}
}

char * GpuLengthSlidingWindowFilterKernel::GetResultEventBuffer()
{
	return p_ResultEventBuffer->GetHostEventBuffer();
}

int GpuLengthSlidingWindowFilterKernel::GetResultEventBufferSize()
{
	return p_ResultEventBuffer->GetEventBufferSizeInBytes();
}

}

#endif
